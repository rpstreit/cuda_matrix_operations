#include "hip/hip_runtime.h"
/*
void vectormagnitude() {
    return sqrt((x2-x1)^2 + (y2-y1)^2)
}


void matrixMultiple() {
    
}

void matrixTranspose() {
    
}
*/
#include "../include/matrix.h"
#include <iostream>

#define THREADS_PER_BLOCK 256



////////////////////////////
// CUDA Functions Section //
////////////////////////////

/**
 * Sum of a vector, taken from Dr. Garg's GitHub
 * @param d_out vector output
 * @param d_in  vector input
 */
__global__ void global_reduce_add_kernel(float * d_out, float * d_in)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // do reduction in global mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            d_in[myId] += d_in[myId + s];
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = d_in[myId];
    }
}

/**
 * CUDA parallel Dot Product (kinda. It just calculates the vector for adding later)
 * @param product the returned dot product
 * @param vector1 first vector
 * @param vector2 second vector
 * @param size    size of the vectors
 */
__global__ void kindaDotProduct(double *product, double* vector1, double* vector2, double size) {
    int idx = threadIDx.x + blockIdx.x * blockDim.x;
    if(idx < size)
        product[idx] = vector1[idx] * vector2[idx];
}


///////////////////////////////
// General Functions Section //
///////////////////////////////

/**
 * Does the dot product of 2 vectors
 * @param vector1   first vector
 * @param vector2   second vector
 * @return          dot product of the vectors....
 */
double dotProduct(linalg::Matrix& vector1, linalg::Matrix& vector2) {
    // CUDA setup
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
    {
    std::cerr << "error: no devices supporting CUDA" << std::endl;
    exit(EXIT_FAILURE);
    }

    int device = 0;
    hipSetDevice(device);

    // Error checking
    // One of rows or columns must be 1 to be a vector
    int numRows1 = vector1.getNumRows();
    int numCols1 = vector1.getNumCols();
    int numRows2 = vector2.getNumRows();
    int numCols2 = vector2.getNumCols();
    if((numRows1 != 1 && numCols1 != 1) || (numRows2 != 1 && numCols2 != 1)) {
        std::cout << "Input to dotProduct is not a vector" << std::endl;
        return -1337;
    }
    
    // Find the size of the vectors, assuming they're the same size
    double size;
    if(numRows1 == 1) {
        size = numCols1 * sizeof(double);
    } else {
        size = numRows1 * sizeof(double);
    }
    // TODO vectors must be the same size

    double **base1 = vector1.GetRaw();
    double **base2 = vector2.GetRaw();

    double *d_vector1;
    double *d_vector2;
    double *d_intermediate;
    // Allocate and copy the stuff to the device
    hipMalloc((void **) &d_vector1, size);
    hipMalloc((void **) &d_vector2, size);
    hipMalloc((void **) &d_intermediate, size);
    hipMemcpy(d_vector1, *base1, size, hipMemcpyHostToDevice);
    hipMemcpy(d_vector2, *base2, size, hipMemcpyHostToDevice);

    // Call CUDA function
    kindaDotProduct<<<(size / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(d_intermediate, d_vector1, d_vector2, size);

    int threads = THREADS_PER_BLOCK;
    int blocks = (size / THREADS_PER_BLOCK);

    // Call CUDA function for summing
    // I reuse d_vector1 because I'm lazy
    global_reduce_add_kernel<<<blocks, threads>>>(d_vector1, d_intermediate);
    
    // Call CUDA function for summing part 2
    threads = blocks;
    blocks = 1;
    global_reduce_add_kernel<<<blocks, threads>>>(d_intermediate, d_vector1);


    double product;
    // Copy over answer
    hipMemcpy(&product, d_intermediate, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_vector1);
    hipFree(d_vector2);
    hipFree(d_intermediate);

    // Return product
    return product;
} 



linalg::Matrix transpose(linalg::Matrix& matrix) {
    // CUDA setup
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
    {
    std::cerr << "error: no devices supporting CUDA" << std::endl;
    exit(EXIT_FAILURE);
    }

    int device = 0;
    hipSetDevice(device);
}

