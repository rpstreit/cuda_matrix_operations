#include "hip/hip_runtime.h"
#include "common.h"
#include <vector>

Matrix * steepestDescent(Matrix *A_operator, Matrix *b_operator) {
    const double error = .005;
    double size = b_operator->GetNumRows();
    Matrix *x0 = new Matrix(size, 0); // make a column vector
    Matrix *x_current = x0;

    Matrix *x_next;
    Matrix *d_vector;
    // Allocate all intermediate matrices required
    Matrix * A_xk_1 = new Matrix(A_operator->GetNumRows(), 1);
    Matrix * A_dk_1 = new Matrix(A_operator->GetNumRows(), 1);
    Matrix * dk_1_A_dk_1 = new Matrix(1, 1);

    do {
        // Find dk-1
        // dk-1 = -1 * (A * xk-1 - b)
        matrix_multiply(A_operator, x_current, A_xk_1);
        
        // Note that this creates a new matrix
        d_vector = &((*A_xk_1 - *b_operator) * -1);
        
        double dk_1sq = pow(norm(d_vector), 2);

        matrix_multiply(A_operator, d_vector, A_dk_1);
        double denominator = dot_product(d_vector, A_dk_1, dk_1_A_dk_1);    

        x_next = &(*x_current + (*d_vector * (dk_1sq / denominator)));
        delete x_current;
        x_current = x_next;
    } while(norm(d_vector) > error);

    delete A_xk_1;
    delete A_dk_1;
    delete dk_1_A_dk_1;
    delete d_vector;

    return x_current;
}

std::vector<Matrix *> constructAConjugates(Matrix * A_operator) {
    // Assuming b_operator is a column vector
    std::vector<Matrix *> p_vectors;
    int length = A_operator->GetNumRows();
    // Matrix *p0 = new Matrix(length, 1);
    // (*p0)[0][0] = 1;
    // p_vectors.push_back(p0);


    // Create a linearly independent set
    // The easiest set turns out to be essentially an identity matrix
    for(int i=0; i<length; i++) {
        Matrix *temp = new Matrix(length, 1);
        (*temp)[i][0] = 1;
        p_vectors.push_back(temp);
    }


    Matrix *pj_t = new Matrix(1, length);
    Matrix *pj_t_A = new Matrix(1, length);
    Matrix *pj_t_A_pk = new Matrix(1, 1);
    Matrix *pj_t_A_pj = new Matrix(1, 1);

    for(int k=1; k<length; k++) {
        for(int j=0; j<k; j++) {
            // get Pj transpose
            matrix_transpose(p_vectors[j], pj_t);
            
            // Get Pj transpose * A
            matrix_multiply(pj_t, A_operator, pj_t_A);

            // Get (Pj transpose * A) * Pk
            matrix_multiply(pj_t_A, p_vectors[k], pj_t_A_pk);

            // final value of the numerator
            double numerator = (*pj_t_A_pk)[0][0];

            // Get (Pj transpose * A) * Pj
            matrix_multiply(pj_t_A, p_vectors[k], pj_t_A_pj);
            
            // final value of the denominator
            double denominator = (*pj_t_A_pj)[0][0];

            double multiplier = numerator / denominator;
            // TODO does this delete properly?
            p_vectors[k] = &(*(p_vectors[k]) - (*(p_vectors[j]) * multiplier)); 
        }
    }

    delete pj_t;
    delete pj_t_A;
    delete pj_t_A_pk;
    delete pj_t_A_pj;

    return p_vectors;
}


Matrix * conjugateDirection(Matrix * A_operator, Matrix * b_operator) {
    Matrix *x0 = new Matrix(A_operator->GetNumRows(), 0); // make a column vector
    Matrix *xcurrent; // make a column vector
    std::vector<Matrix *> A_conjugates = constructAConjugates(A_operator);
    double ak;
    // we will have our guess of x0 be 0 so that r0 = b
    
    Matrix * pk_t = new Matrix(1, A_conjugates[0]->GetNumRows());
    Matrix * pk_t_r0 = new Matrix(1, 1);
    Matrix * A_pk = new Matrix(A_operator->GetNumCols(), b_operator->GetNumRows());
    Matrix * pk_t_A_pk = new Matrix(1, 1);
    // Limited run
    for(int k=0; k < A_operator->GetNumRows(); k++) {
        matrix_transpose(A_conjugates[k], pk_t);
        matrix_multiply(pk_t, b_operator, pk_t_r0);
        int numerator = (*pk_t_r0)[0][0];

        matrix_multiply(A_operator, A_conjugates[k], A_pk);
        matrix_multiply(pk_t, A_pk, pk_t_A_pk);
        ak = (*pk_t_A_pk)[0][0];
        xcurrent = &(*x0 + (*(A_conjugates[k]) * ak));
        delete x0;
        x0 = xcurrent;
    }

    delete pk_t;
    delete pk_t_r0;
    delete A_pk;
    delete pk_t_A_pk;

    for(int i=0; i<A_conjugates.size(); i++) {
        delete A_conjugates[i];
    }
    return x0;
}
