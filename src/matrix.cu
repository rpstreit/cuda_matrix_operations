#include "hip/hip_runtime.h"

#include <fstream>

#include "matrix.h"
#include "common.h"

__global__ void kset_zeroes(double *A);
__global__ void kset_identity(double *A, int cols);

Matrix::Matrix(const char *file) :
  num_rows(0),
  num_cols(0),
  flat(0)
{
  this->Parse(file);
}

Matrix::Matrix(const Matrix &copy) :
  num_rows(copy.num_rows),
  num_cols(copy.num_cols),
  flat(0)
{
  hipMallocManaged(&flat, sizeof(double) * copy.num_rows * copy.num_cols);
  for (int i = 0; i < this->num_rows; ++i)
  {
    for (int j = 0; j < this->num_cols; ++j)
    {
      this->flat[i * copy.num_cols + j] = copy.flat[i * copy.num_cols + j];
    }
  }
}

Matrix::Matrix(int num_rows, int num_cols, bool identity) :
  num_rows(num_rows),
  num_cols(num_cols),
  flat(0)
{
  hipMallocManaged(&flat, sizeof(double) * num_rows * num_cols);

  if (identity)
  {
    ToIdentity();
  }
}

Matrix::~Matrix(void)
{ 
  hipFree(this->flat);
}


/////////////////////
// CUDA Operations //
/////////////////////
__global__ void kset_identity(Matrix *A)
{ 
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  
  if ((idx / A->GetNumCols()) == (idx % A->GetNumCols()))
  {
    A->GetFlattened()[idx] = 1;
  }
  else
  {
    A->GetFlattened()[idx] = 0;
  }
}

__global__ void ksubtract(Matrix *output, Matrix *A, Matrix *B)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < (A->GetNumCols() * A->GetNumRows())) 
  {
    int row = idx / A->GetNumCols();
    int col = idx % A->GetNumCols();
    (*output)[row][col] = (*A)[row][col] - (*B)[row][col];
  }
}

__global__ void kadd(Matrix *output, Matrix *A, Matrix *B)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < (A->GetNumCols() * A->GetNumRows())) 
  {
    int row = idx / A->GetNumCols();
    int col = idx % A->GetNumCols();
    (*output)[row][col] = (*A)[row][col] + (*B)[row][col];
  }
}

__global__ void kmultiply(Matrix *output, Matrix *input, double scale)
{
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if(idx < (input->GetNumCols() * input->GetNumRows())) 
  {
    int row = idx / input->GetNumCols();
    int col = idx % input->GetNumCols();
    (*output)[row][col] = (*input)[row][col] * scale;
  }
}


void Matrix::ToZeroes(void)
{
  int num_blocks = (num_cols * num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kset_zeroes<<<num_blocks, THREADS_PER_BLOCK>>>(this->flat);
  hipDeviceSynchronize();
}

void Matrix::ToIdentity(void)
{
  int num_blocks = (num_cols * num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kset_identity<<<num_blocks, THREADS_PER_BLOCK>>>(this->flat, this->num_cols);
  hipDeviceSynchronize();
}

////////////////////////
// Operator Overloads //
////////////////////////
__host__ __device__ double * Matrix::operator[](int row_idx)
{
  return &(this->flat[row_idx * this->num_cols]);
}

Matrix& Matrix::operator-(Matrix& other) 
{
  Matrix *blah = new Matrix(this->num_rows, this->num_cols);
  int num_blocks = (this->num_cols * this->num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  ksubtract<<<num_blocks, THREADS_PER_BLOCK>>>(blah, this, &other);
  return *blah;
}

Matrix& Matrix::operator+(Matrix& other) 
{
  Matrix *blah = new Matrix(this->num_rows, this->num_cols);
  int num_blocks = (this->num_cols * this->num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kadd<<<num_blocks, THREADS_PER_BLOCK>>>(blah, this, &other);
  return *blah;
}

Matrix& Matrix::operator*(double scale) 
{
  Matrix *blah = new Matrix(this->num_rows, this->num_cols);
  int num_blocks = (this->num_cols * this->num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kmultiply<<<num_blocks, THREADS_PER_BLOCK>>>(blah, this, scale);
  return blah;
}



void Matrix::Parse(const char* file)
{
  std::ifstream matrix(file);

  if (this->flat != 0)
  {
    hipFree(this->flat);
  }

  matrix >> this->num_rows;
  matrix >> this->num_cols;

  hipMallocManaged(&flat, sizeof(double) * num_rows * num_cols);
  for (int i = 0; i < this->num_rows; ++i)
  {
    for (int j = 0; j < this->num_cols; ++j)
    {
      matrix >> this->flat[i * this->num_cols + j];
    }
  }
}


__host__ __device__ double & Matrix::At(int row, int col)
{
  return (*this)[row][col];
}

__host__ __device__ double * Matrix::GetFlattened(void)
{
  return this->flat;
}

__host__ __device__ int Matrix::GetNumCols(void)
{
  return this->num_cols;
}

__host__ __device__ int Matrix::GetNumRows(void)
{
  return this->num_rows;
}

__global__ void kset_zeroes(double *A)
{ 
 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
  A[idx] = 0;
}

__global__ void kset_identity(double *A, int cols)
{ 
 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
  if ((idx / cols) == (idx % cols))
  {
    A[idx] = 1;
  }
  else
  {
    A[idx] = 0;
  }
}

