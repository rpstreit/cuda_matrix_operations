#include "hip/hip_runtime.h"

#include <fstream>

#include "matrix.h"

Matrix::Matrix(const char *file)
{
  this->Parse(file);
}

Matrix::Matrix(const Matrix &copy) :
  num_rows(copy.num_rows),
  num_cols(copy.num_cols)
{
  this->flat = hipMallocManaged(&flat, sizeof(double) * copy.num_rows * copy.num_cols);
  for (int i = 0; i < this->num_rows; ++i)
  {
    for (int j = 0; j < this->num_cols; ++j)
    {
      this->flat[i * copy.num_cols + j] = copy.flat[i * copy.num_cols + j];
    }
  }
}

Matrix::Matrix(int num_rows, int num_cols, bool identity) :
  num_rows(num_rows),
  num_cols(num_cols)
{
  this->flat = hipMallocManaged(&flat, sizeof(double) * num_rows * num_cols);

  if (identity)
  {
    int num_blocks = (num_cols * num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    this->set_identity<<<num_blocks, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();
  }
}

Matrix::~Matrix(void)
{ 
  hipFree(this->flat);
}

__global__ Matrix::set_identity(void)
{ 
 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
  if ((idx / num_cols) == (idx % num_cols))
  {
    this->flat[idx] = 1;
  }
  else
  {
    this->flat[idx] = 0;
  }
}

double * operator[](int row_idx)
{
  return &(this-flat[row_idx * this->num_cols]);
}

void Matrix::Parse(const char* file)
{
  ifstream matrix(file);

  if (this->flat != 0)
  {
    hipFree(this->flat);
  }

  matrix >> this->num_rows;
  matrix >> this->num_cols;

  this->flat = hipMallocManaged(&flat, sizeof(double) * copy.num_rows * copy.num_cols);
  for (int i = 0; i < this->num_rows; ++i)
  {
    for (int j = 0; j < this->num_cols; ++j)
    {
      matrix >> this->flat[i * this->num_cols + j];
    }
  }
}

__host__ __device__ double * Matrix::GetFlattened(void)
{
  return this->flat;
}

__host__ __device__ int Matrix::GetNumCols(void)
{
  return this->num_cols;
}

__host__ __device__ int Matrix::GetNumRows(void)
{
  return this->num_rows;
}
