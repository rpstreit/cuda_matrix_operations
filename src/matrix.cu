#include "hip/hip_runtime.h"

#include <fstream>

#include "common.h"
#include "matrix.h"

Matrix::Matrix(const char *file)
{
  this->Parse(file);
}

Matrix::Matrix(const Matrix &copy) :
  num_rows(copy.num_rows),
  num_cols(copy.num_cols)
{
  hipMallocManaged(&flat, sizeof(double) * copy.num_rows * copy.num_cols);
  for (int i = 0; i < this->num_rows; ++i)
  {
    for (int j = 0; j < this->num_cols; ++j)
    {
      this->flat[i * copy.num_cols + j] = copy.flat[i * copy.num_cols + j];
    }
  }
}

Matrix::Matrix(int num_rows, int num_cols, bool identity) :
  num_rows(num_rows),
  num_cols(num_cols)
{
  hipMallocManaged(&flat, sizeof(double) * num_rows * num_cols);

  if (identity)
  {
    set_identity();
  }
}

Matrix::~Matrix(void)
{ 
  hipFree(this->flat);
}

__global__ void kset_identity(Matrix *A)
{ 
 	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	
  if ((idx / A->GetNumCols()) == (idx % A->GetNumCols()))
  {
    A->GetFlattened()[idx] = 1;
  }
  else
  {
    A->GetFlattened()[idx] = 0;
  }
}

void Matrix::set_identity(void)
{
  int num_blocks = (num_cols * num_rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  kset_identity<<<num_blocks, THREADS_PER_BLOCK>>>(this);
  hipDeviceSynchronize();
}

__host__ __device__ double & Matrix::At(int row, int col)
{
  return (*this)[row][col];
}

__host__ __device__ double * Matrix::operator[](int row_idx)
{
  return &(this->flat[row_idx * this->num_cols]);
}

void Matrix::Parse(const char* file)
{
  std::ifstream matrix(file);

  if (this->flat != 0)
  {
    hipFree(this->flat);
  }

  matrix >> this->num_rows;
  matrix >> this->num_cols;

  hipMallocManaged(&flat, sizeof(double) * num_rows * num_cols);
  for (int i = 0; i < this->num_rows; ++i)
  {
    for (int j = 0; j < this->num_cols; ++j)
    {
      matrix >> this->flat[i * this->num_cols + j];
    }
  }
}

__host__ __device__ double * Matrix::GetFlattened(void)
{
  return this->flat;
}

__host__ __device__ int Matrix::GetNumCols(void)
{
  return this->num_cols;
}

__host__ __device__ int Matrix::GetNumRows(void)
{
  return this->num_rows;
}

