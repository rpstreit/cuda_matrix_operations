#include "hip/hip_runtime.h"

#include <iostream>

#include "common.h"
#include "matrix.h"
#include "cpu.h"

// Assumes that A->GetNumRows() >= A->GetNumCols()
void lu_decomposition(Matrix *A, Matrix *L, Matrix *U, Matrix *P)
{
  if (A->GetNumRows() < A->GetNumCols())
  {
    std::cerr << "lu_decomposition: matrix dimensions on A are ill formed for LU Decomposition" << std::endl;
    exit(EXIT_FAILURE);
  }
  if (A->GetNumRows() != L->GetNumRows()
      || A->GetNumRows() != U->GetNumRows()
      || A->GetNumRows() != P->GetNumRows()
      || A->GetNumCols() != U->GetNumCols()
      || L->GetNumRows() != L->GetNumCols()
      || P->GetNumRows() != P->GetNumCols())
  {
    std::cerr << "lu_decomposition: matrix dimensions of inputs are mismatched" << std::endl;
  }
  double *column_slice;
  P->ToIdentity();
  L->ToIdentity();
  matrix_copy(U, A);
  int rows = A->GetNumRows();
  int cols = A->GetNumCols();

  Matrix *U_intermediate = new Matrix(A->GetNumRows(), A->GetNumCols());
  Matrix *P_intermediate = new Matrix(P->GetNumRows(), P->GetNumCols());
  Matrix *P_acc = new Matrix(P->GetNumRows(), P->GetNumCols());
  Matrix *L_intermediate = new Matrix(L->GetNumRows(), L->GetNumCols());
  hipMalloc((void **) &column_slice, sizeof(double) * rows);

  P->ToIdentity(); // O(1)
  for (int i = 0; i < cols - 1; i++)
  {
    matrix_slicecolumn(U, column_slice, i); // O(1)
//    double slice[rows];
//    hipMemcpy(slice, column_slice, rows * sizeof(double), hipMemcpyDeviceToHost);
//    std::cout << "col " << i << " slice:\n{";
//    for (int j = 0; j < rows; ++j)
//    {
//      std::cout << " " << slice[j];
//    }
//    std::cout << " }" << std::endl;
    int idx;
    double max = reduce_absmaxidx(&column_slice[i], rows - i, &idx); // O(log(rows - i)) <= O(log(rows))
    idx = idx + i;
    std::cout << "col " << i << ", max: " << max << "@row " << idx;

    if (i != idx)
    {
      matrix_rowswap(P, i, idx); // O(1)
      matrix_rowswap(U, i, idx);
      matrix_subdiagonal_rowswap(L, i, idx);
    }

    // I reuse pointers in ways that don't match the names below
    // just to save on copies
    // Update U
    matrix_getelementarymatrix(U, L_intermediate, i);
    std::cout << "elementary matrix for col " << i << ": " << std::endl;
    matrix_print(L_intermediate);
    matrix_multiply(L_intermediate, U, U_intermediate);
    matrix_copy(U, U_intermediate);

    // Update L
    matrix_invertelementarymatrix(L_intermediate, P_intermediate, i);
    std::cout << "\ninverted elementary matrix:" << std::endl;
    matrix_print(P_intermediate);

    matrix_subdiagonal_writecolumn(L, P_intermediate, i);

    std::cout << "\nCurr U:" << std::endl;
    matrix_print(U);

    std::cout << "\nCurr P:" << std::endl;
    matrix_print(P);

    std::cout << "\nCurr L:" << std::endl;
    matrix_print(L);
  }
  
  delete U_intermediate;
  delete P_intermediate;
  delete P_acc;
  delete L_intermediate;
  hipFree(column_slice);
}

void lu_blockeddecomposition(Matrix *A, Matrix *L, Matrix *U, Matrix *P)
{

}
