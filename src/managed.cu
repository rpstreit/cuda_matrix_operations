
#include <cstdio>
#include "managed.h"
void * Managed::operator new(size_t len)
{
  void *ptr;
  gpuErrchk(hipMallocManaged(&ptr, len));
  gpuErrchk(hipDeviceSynchronize());
  return ptr;
}

void Managed::operator delete(void *ptr)
{
  hipDeviceSynchronize();
  hipFree(ptr);
}
