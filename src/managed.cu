
#include <cstdio>
#include "managed.h"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
void * Managed::operator new(size_t len)
{
  void *ptr;
  gpuErrchk(hipMallocManaged(&ptr, len));
  gpuErrchk(hipDeviceSynchronize());
  return ptr;
}

void Managed::operator delete(void *ptr)
{
  hipDeviceSynchronize();
  hipFree(ptr);
}
