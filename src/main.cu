#include "hip/hip_runtime.h"

#include <cstring>
#include <iostream>

#include "common.h"
#include "matrix.h"
#include "lu_decomposition.h"
#include "tests.h"

#define VERIFY_KEY "verify"

typedef int (*routine_t)(Matrix *A, ...);

struct operation_t
{
  char const * name; // name expected from command line
  int num_args; // number of matrices to pass to this operation
  routine_t run; // routine to do operation and print result to 
                   // stdout. Return 0 on success (if there is any
                   // reason to fail)
  routine_t verify; // routine to test operation. On success return 0,
                  // otherwise if the test fails, return something else
};

enum Operations
{
  MATMUL,
 
  // do not enter anything else after here

  COUNT
};

operation_t ops[COUNT] =
{
  {"matmul", 2, matmul_run, matmul_verify},
};

int main(int argc, char **argv)
{  
  if (argc < 2)
  {
    std::cerr << "error: no operation specified!" << std::endl;
    exit(EXIT_FAILURE);
  }

  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0)
  {
    std::cerr << "error: no devices supporting CUDA" << std::endl;
    exit(EXIT_FAILURE);
  }
  
  int device = 0;
  hipSetDevice(device);

  hipDeviceProp_t deviceProps;
  if (!hipGetDeviceProperties(&deviceProps, device))
  {
    std::cout << "Using device " << device << ":" << std::endl;
    std::cout << deviceProps.name << "; global mem: " << deviceProps.totalGlobalMem
      << "; compute v" << deviceProps.major << "." << deviceProps.minor << 
      "; clock: " << deviceProps.clockRate << "kHz" << std::endl;
  } 

  // iterate through test array and execute matching name
}
