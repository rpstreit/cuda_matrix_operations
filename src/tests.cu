
#include <cstdarg>
#include <iostream>

#include "tests.h"
#include "common.h"
#include "cpu.h"
#include "matrix.h"
#include "lu_decomposition.h"
#include "linearSysSolver.h"

int matmul_run(int argc, Matrix **argv) 
{
  if (argc != 2)
  {
    std::cerr << "error: matmul requires 2 arguments" << std::endl;
    exit(EXIT_FAILURE);
  }

  Matrix *A = argv[0];
  Matrix *B = argv[1];
  if (A->GetNumCols() != B->GetNumRows())
  {
    std::cerr << "error: num cols in input 1 does not equal num rows of input 2" << std::endl;
    exit(EXIT_FAILURE);
  }
  Matrix *result = new Matrix(A->GetNumRows(), B->GetNumCols());
  
  matrix_multiply(A, B, result);
  std::cout << "\nAB = " << std::endl;
  matrix_print(result);

  delete result;
  return 0;
}

int matmul_verify(int argc, Matrix **argv)
{
  if (argc != 2)
  {
    std::cerr << "error: matmul requires 2 arguments" << std::endl;
    exit(EXIT_FAILURE);
  }
  
  Matrix *A = argv[0];
  Matrix *B = argv[1];
  if (A->GetNumCols() != B->GetNumRows())
  {
    std::cerr << "error: num cols in input 1 does not equal num rows of input 2" << std::endl;
    exit(EXIT_FAILURE);
  }

  Matrix *result_cpu = new Matrix(A->GetNumRows(), B->GetNumCols());
  Matrix *result_gpu = new Matrix(A->GetNumRows(), B->GetNumCols());
  
  matrix_multiply_cpu(A, B, result_cpu);
  matrix_multiply(A, B, result_gpu);

  hipDeviceSynchronize();
  int result = matrix_equals(result_cpu, result_gpu, ERROR) ? 0 : 1;

  hipDeviceSynchronize();
  delete result_cpu;
  delete result_gpu;

  return result;
}

int lu_decomposition_run(int argc, Matrix **argv)
{
  if (argc != 1)
  {
    std::cerr << "error: lu decomposition requires 1 argument" << std::endl;
  }

  Matrix *A = argv[0];
  Matrix *P = new Matrix(A->GetNumCols(), A->GetNumCols());
  Matrix *L = new Matrix(A->GetNumCols(), A->GetNumCols());
  Matrix *U = new Matrix(A->GetNumRows(), A->GetNumCols());
  Matrix *left = new Matrix(A->GetNumRows(), A->GetNumCols());
  Matrix *right = new Matrix(A->GetNumRows(), A->GetNumCols());

  lu_decomposition(A, L, U, P);

  std::cout << "\nP =" << std::endl;
  matrix_print(P);
  std::cout << "\nL =" << std::endl;
  matrix_print(L);
  std::cout << "\nU =" << std::endl;
  matrix_print(U);

  delete P;
  delete L;
  delete U;
  delete left;
  delete right;

  return 0;
}

int lu_decomposition_verify(int argc, Matrix **argv)
{
  if (argc != 1)
  {
    std::cerr << "error: lu decomposition requires 1 argument" << std::endl;
  }

  Matrix *A = argv[0];
  Matrix *P = new Matrix(A->GetNumCols(), A->GetNumCols());
  Matrix *L = new Matrix(A->GetNumCols(), A->GetNumCols());
  Matrix *U = new Matrix(A->GetNumRows(), A->GetNumCols());
  Matrix *left = new Matrix(A->GetNumRows(), A->GetNumCols());
  Matrix *right = new Matrix(A->GetNumRows(), A->GetNumCols());

  lu_decomposition(A, L, U, P);

  matrix_multiply_cpu(P, A, left);
  matrix_multiply_cpu(L, U, right);

  std::cout << "\nPA = " << std::endl;
  matrix_print(left);
  std::cout << "\nLU = " << std::endl;
  matrix_print(right);
  int result = matrix_equals(left, right, ERROR) ? 0 : 1;

  delete P;
  delete L;
  delete U;
  delete left;
  delete right;

  return result;
}

int linear_descent_run(int argc, Matrix **argv)
{
  if(argc != 2)
  {
    std::cerr << "error: lu decomposition requires 1 argument" << std::endl;
  }
  Matrix * A_operator = argv[0];
  matrix_print(A_operator);
  Matrix * b_operator = argv[1];
  matrix_print(b_operator);

  Matrix * output = steepestDescent(A_operator, b_operator);
  matrix_print(output);

  delete A_operator;
  delete b_operator;
  delete output;

  return 0;
}

int conjugate_direction_run(int argc, Matrix **argv)
{
  if(argc != 2)
  {
    std::cerr << "error: lu decomposition requires 1 argument" << std::endl;
  }
  
  Matrix * A_operator = argv[0];
  matrix_print(A_operator);
  Matrix * b_operator = argv[1];
  matrix_print(b_operator);

  Matrix * output = conjugateDirection(A_operator, b_operator);
  matrix_print(output);

  delete A_operator;
  delete b_operator;
  delete output;

  return 0;
}

int determinant_recur_run(int argc, Matrix **argv)
{
  if(argc != 1)
  {
    std::cerr << "error: lu decomposition requires 1 argument" << std::endl;
  }
  Matrix * A_operator = argv[0];
  matrix_print(A_operator);

  int determinant = determinant_recur(A_operator);
  std::cout << determinant << std::endl;

  delete A_operator;
}

int linear_solve_verify(int argc, Matrix **argv)
{
  return 0;
}

int determinant_verify(int argc, Matrix **argv)
{
  return 0;
}
